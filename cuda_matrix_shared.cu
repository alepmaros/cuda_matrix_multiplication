
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

#define NLINES 6144
#define NCOLUMNS 6144
#define THREADS_PER_BLOCK 1024
//32x32
#define NTHREADS 32
#define NCOLUMNSPERBLOCK 32
#define NLINESPERBLOCK 32

__global__ void vector_mul(int *a, int *b, int *c) {
    int i, z, sum = 0;

    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int line =  blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ int s_a[NLINESPERBLOCK][NCOLUMNSPERBLOCK];
    __shared__ int s_b[NLINESPERBLOCK][NCOLUMNSPERBLOCK];

	for (z = 0; z < gridDim.x; z++) {
        s_a[threadIdx.y][threadIdx.x] = a[ (NCOLUMNS * (blockIdx.y * NLINESPERBLOCK + threadIdx.y)) + (z * NCOLUMNSPERBLOCK + threadIdx.x) ];
        s_b[threadIdx.y][threadIdx.x] = b[ (NCOLUMNS * (z * NLINESPERBLOCK + threadIdx.y)) + blockIdx.x * NCOLUMNSPERBLOCK + threadIdx.x ];

        __syncthreads();

		for (i = 0; i < NLINESPERBLOCK; i++) {
	    	sum += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
	    }

        __syncthreads();
	}

    //printf("%d %d\n", line, column);
    c[line * NLINES + column] = sum;
}

int main(){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = NLINES * NCOLUMNS * sizeof(int);
    int i, j, n;

    struct timeval timevalA;
	struct timeval timevalB;

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for(i = 0; i < NLINES*NCOLUMNS; i++){
        c[i] = 0;
    }

    scanf("%d", &n);

    for (int i = 0; i < NLINES; i++) {
        for (j = 0; j < NLINES; j++) {
            scanf("%d", &a[i * NLINES + j]);
        }
    }

    for (int i = 0; i < NLINES; i++) {
        for (j = 0; j < NLINES; j++) {
            scanf("%d", &b[i * NLINES + j]);
        }
    }

    gettimeofday(&timevalA,NULL);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 tbloco = dim3(NLINES/NTHREADS, NLINES/NTHREADS,1);
    dim3 tthreads = dim3(NTHREADS, NTHREADS, 1);
    vector_mul<<<tbloco,tthreads>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    gettimeofday(&timevalB,NULL);

    // print Matrix
    // for (i = 0; i < NLINES; i++) {
    //     for (j = 0; j < NLINES; j++) {
    //         printf("%d ", c[i * NLINES + j]);
    //     }
    //     printf("\n");
    // }
    //printf("\n");

    printf("%.5lf\n", timevalB.tv_sec-timevalA.tv_sec+(timevalB.tv_usec-timevalA.tv_usec)/(double)1000000);

    free(a); free(b); free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}